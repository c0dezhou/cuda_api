#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <atomic>
#include <thread>
#include <math.h>

__global__ void dummyKernel() {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx == 0) {
        float result = 0.0f;
        for (int i = 0; i < 1000000; ++i) {
            result += sinf(static_cast<float>(i)) * cosf(static_cast<float>(i));
        }
    }
}

void createAndDestroyEvent(hipEvent_t* event, std::atomic<bool>& done) {
    hipEventCreate(event);
    dummyKernel<<<1, 1>>>();
    hipEventRecord(*event, 0);

    std::this_thread::sleep_for(std::chrono::milliseconds(100));
    hipEventDestroy(*event);
    done.store(true);
}

void waitForEvent(hipEvent_t* event,
                  std::atomic<bool>& done,
                  hipError_t* result) {
    while (!done.load()) {
        *result = hipEventSynchronize(*event);
        if (*result != hipSuccess) {
            break;
        }
    }
}

void multithreadedEventDestruction() {
    std::atomic<bool> done(false);
    hipEvent_t event;
    hipError_t waitForEventResult;

    // Create two threads, one for creating and destroying events, and another
    // for waiting on events
    std::thread createAndDestroyThread(createAndDestroyEvent, &event,
                                       std::ref(done));
    std::thread waitForEventThread(waitForEvent, &event, std::ref(done),
                                   &waitForEventResult);

    createAndDestroyThread.join();
    waitForEventThread.join();


    // Check if hipEventSynchronize returned an error
    EXPECT_NE(waitForEventResult, hipSuccess);
}

void event_other_thread_use() {
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    dummyKernel<<<1, 1, 0, stream1>>>();
    hipEventRecord(event1, stream1);
    dummyKernel<<<1, 1, 0, stream2>>>();
    hipEventRecord(event2, stream2);

    // Add hipStreamWaitEvent to make stream2 wait for event1
    hipStreamWaitEvent(stream2, event1, 0);

    // Wait until both streams are actively executing the kernels
    while (hipStreamQuery(stream2) == hipErrorNotReady) {

        hipError_t result1 = hipEventDestroy(event1);
        hipError_t result2 = hipEventDestroy(event2);

        EXPECT_NE(result1, hipSuccess);
        EXPECT_NE(result2, hipSuccess);
    }

    

    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

}